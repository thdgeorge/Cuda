/*
* This sample implements a separable convolution
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

//unsigned int filter_radius;

#define filter_radius 16
#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.00005
#define TILE_WIDTH 32
#define TILE_W1 64
#define TILE_H1 64
#define TILE_W2 64
#define TILE_H2 64


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter, int imageW, int imageH, int filterR);
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter, int imageW, int imageH, int filterR);

__constant__ float d_Filter[FILTER_LENGTH];

//Kernel gia efarmogh filtrou kata grammes
 __global__
 void convolutionRowGPU(float *d_Buffer, float *d_Input, int imageW, int imageH) {

	__shared__ float data[TILE_H1 * (TILE_W1 + 2*filter_radius)];

	int y = blockIdx.y * 2*blockDim.y + threadIdx.y;
	int x = blockIdx.x * 2*blockDim.x + threadIdx.x;

	int reg = (y + filter_radius) * (imageW + 2*filter_radius) + (x + filter_radius);
	int data_reg = threadIdx.x + threadIdx.y * (TILE_W1 + 2*filter_radius);

	int k;
	data[data_reg] = d_Input[reg - filter_radius];
	data[data_reg + blockDim.x] = d_Input[reg + filter_radius];
	data[data_reg + 2*blockDim.x] = d_Input[reg + filter_radius + blockDim.x];
	
	data[data_reg + blockDim.y * (TILE_W1 + 2*filter_radius)] = d_Input[reg + blockDim.y * (imageW + 2*filter_radius) - filter_radius];
	data[data_reg + blockDim.x + blockDim.y * (TILE_W1 + 2*filter_radius)] = d_Input[reg + blockDim.y * (imageW + 2*filter_radius) + filter_radius];
	data[data_reg + 2*blockDim.x + blockDim.y * (TILE_W1 + 2*filter_radius)] = d_Input[reg + blockDim.y * (imageW + 2*filter_radius) + filter_radius + blockDim.x];
	

	__syncthreads();

	float sum1, sum2, sum3, sum4;
	sum1 = sum2 = sum3 = sum4 = 0;
	#pragma unroll
	for (k = -filter_radius; k <= filter_radius; k++) {
		sum1 += data[data_reg + filter_radius + k] * d_Filter[filter_radius - k];
		sum2 += data[data_reg + blockDim.x + filter_radius + k] * d_Filter[filter_radius - k];
		sum3 += data[data_reg + blockDim.y * (TILE_W1 + 2*filter_radius) + filter_radius + k] * d_Filter[filter_radius - k];
		sum4 += data[data_reg + blockDim.y * (TILE_W1 + 2*filter_radius) + blockDim.x + filter_radius + k] * d_Filter[filter_radius - k];
	}

	d_Buffer[reg] = sum1;
	d_Buffer[reg + blockDim.x] = sum2;
	d_Buffer[reg + blockDim.y * (imageW + 2*filter_radius)] = sum3;
	d_Buffer[reg + blockDim.y * (imageW + 2*filter_radius) + blockDim.x] = sum4;
 }

//Kernel gia efarmogh filtrou kata sthles
 __global__
 void convolutionColumnGPU(float *d_Output, float *d_Buffer, int imageW, int imageH) {

	__shared__ float data[TILE_W2 * (TILE_H2 + 2*filter_radius)];

	int y = blockIdx.y * 2*blockDim.y + threadIdx.y;
	int x = blockIdx.x * 2*blockDim.x + threadIdx.x;

	int reg = (y + filter_radius) * (imageW + 2*filter_radius) + (x + filter_radius);
	int data_reg = threadIdx.x + threadIdx.y * TILE_W2;
	
	data[data_reg] = d_Buffer[reg - (imageW + 2*filter_radius) * filter_radius];
	data[data_reg + blockDim.x] = d_Buffer[reg + blockDim.x - (imageW + 2*filter_radius) * filter_radius];
	
	data[data_reg + blockDim.y * TILE_W2] = d_Buffer[reg + (imageW + 2*filter_radius) * filter_radius];
	data[data_reg + blockDim.y * TILE_W2 + blockDim.x] = d_Buffer[reg + blockDim.x + (imageW + 2*filter_radius) * filter_radius];
	
	data[data_reg + 2*blockDim.y * TILE_W2] = d_Buffer[reg + (imageW + 2*filter_radius) * (filter_radius + blockDim.x)];
	data[data_reg + 2*blockDim.y * TILE_W2 + blockDim.x] = d_Buffer[reg + blockDim.x + (imageW + 2*filter_radius) * (filter_radius + blockDim.x)];


	__syncthreads();

	float sum1, sum2, sum3, sum4;
	sum1 = sum2 = sum3 = sum4 = 0;
	#pragma unroll
	for (int k = -filter_radius; k <= filter_radius; k++) {
		sum1 += data[data_reg + (k + filter_radius) * TILE_W2] * d_Filter[filter_radius - k];
		sum2 += data[data_reg + (k + filter_radius) * TILE_W2 + blockDim.x] * d_Filter[filter_radius - k];
		sum3 += data[data_reg + (blockDim.y + k + filter_radius) * TILE_W2] * d_Filter[filter_radius - k];
		sum4 += data[data_reg + (blockDim.y + k + filter_radius) * TILE_W2 + blockDim.x] * d_Filter[filter_radius - k];
	}

	d_Output[y * imageW + x] = sum1;
	d_Output[y * imageW + x + blockDim.x] = sum2;
	d_Output[(y + blockDim.y) * imageW + x] = sum3;
	d_Output[(y + blockDim.y) * imageW + x + blockDim.x] = sum4;
}

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(float *h_Dst, float *h_Src, float *h_Filter,
                       int imageW, int imageH, int filterR) {

  int x, y, k;

  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = x + k;

        if (d >= 0 && d < imageW) {
          sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
        }

        h_Dst[y * imageW + x] = sum;
      }
    }
  }

}


////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(float *h_Dst, float *h_Src, float *h_Filter,
    			   int imageW, int imageH, int filterR) {

  int x, y, k;

  for (y = 0; y < imageH; y++) {
    for (x = 0; x < imageW; x++) {
      float sum = 0;

      for (k = -filterR; k <= filterR; k++) {
        int d = y + k;

        if (d >= 0 && d < imageH) {
          sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
        }

        h_Dst[y * imageW + x] = sum;
      }
    }
  }

}


////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
	
	//cudaDeviceSetCacheConfig(cudaFuncCachePreferShared);
	//cudaFuncSetCacheConfig(convolutionRowGPU, cudaFuncCachePreferShared);
	//cudaFuncSetCacheConfig(convolutionColumnGPU, cudaFuncCachePreferShared);

	//struct timespec  tv1, tv2;

    float
    *h_Filter,
    *h_Input,
    *h_Buffer,
    //*h_OutputCPU,
	*h_OutputGPU,
	*h_Input_padding,
	*d_Input,
	*d_Buffer,
	*d_Output;


    int imageW;
    int imageH;
    unsigned int i;

	//printf("Enter filter radius : ");
	//scanf("%d", &filter_radius);

    // Ta imageW, imageH ta dinei o xrhsths kai thewroume oti einai isa,
    // dhladh imageW = imageH = N, opou to N to dinei o xrhsths.
    // Gia aplothta thewroume tetragwnikes eikones.
	
	FILE *ifp, *ofp;
    ifp = fopen("Input.txt","r");
    ofp = fopen("Output_32_64x64_64x64.txt","w");
	
	int tile_width = TILE_WIDTH;
	int tile_w1 = TILE_W1;
	int tile_h1 = TILE_H1;
	int tile_w2 = TILE_W2;
	int tile_h2 = TILE_H2;
	
	fprintf(ofp,"TILE_WIDTH: %d\nTILE_W1: %d\nTILE_H1: %d\nTILE_W2: %d\nTILE_H2: %d\n-------------------\n", tile_width, tile_w1, tile_h1, tile_w2, tile_h2);
	

    for (int k = 1; k < 9 ; k++) {

		float sum_GPU = 0;

		//printf("Enter image size. Should be a power of two and greater than %d : ", FILTER_LENGTH);
		fscanf(ifp, "%d", &imageW);
		if ( imageW < FILTER_LENGTH ) {
			printf("Image size lower than %d\nTermatismos programmatos...\n", FILTER_LENGTH);
			return(2);
		}
		else if ( imageW % 2 != 0 ) {
			printf("Image size is not a power of two\nTermatismos programmatos...\n");
			return(3);
		}
		imageH = imageW;

		printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
		
		for (int j = 1; j < 13; j++) {
			
			//printf("Allocating and initializing host arrays...\n");
			// Tha htan kalh idea na elegxete kai to apotelesma twn malloc...
			h_Filter    = (float *)malloc(FILTER_LENGTH * sizeof(float));
			h_Input     = (float *)malloc(imageW * imageH * sizeof(float));
			h_Buffer    = (float *)malloc(imageW * imageH * sizeof(float));
			//h_OutputCPU = (float *)malloc(imageW * imageH * sizeof(float));
			h_OutputGPU = (float *)malloc(imageW * imageH * sizeof(float));
			
			h_Input_padding 	= (float *)malloc((imageW + 2*filter_radius) * (imageH + 2*filter_radius) * sizeof(float));
			
			//Elegxos apotelesmatwn twn malloc
			if (h_Filter == NULL || h_Input == NULL || h_Buffer == NULL || h_OutputGPU == NULL || h_Input_padding == NULL ) {
				printf("Apotuxia Desmeushs mnhmhs \n Termatismos programmatos...\n");
				return(1);
			}

			//Desmeush mnhmhs gia to device
			//printf("Allocating and initializing device arrays...\n");
			gpuErrchk( hipMalloc((void**)&d_Input, (imageW + 2*filter_radius) * (imageH + 2*filter_radius) * sizeof(float)) );
			gpuErrchk( hipMalloc((void**)&d_Buffer, (imageW + 2*filter_radius) * (imageH + 2*filter_radius) * sizeof(float)) );
			gpuErrchk( hipMalloc((void**)&d_Output, imageW * imageH * sizeof(float)) );



			// to 'h_Filter' apotelei to filtro me to opoio ginetai to convolution kai
			// arxikopoieitai tuxaia. To 'h_Input' einai h eikona panw sthn opoia ginetai
			// to convolution kai arxikopoieitai kai auth tuxaia.

			srand(200);

			for (i = 0; i < FILTER_LENGTH; i++) {
				h_Filter[i] = (float)(rand() % 16);
			}

			for (i = 0; i < imageW * imageH; i++) {
				h_Input[i] = (float)rand() / ((float)RAND_MAX / 255) + (float)rand() / (float)RAND_MAX;
			}
			
			
			//Topothethsh tou padding perimetrika ths eikonas eisodou
			for (i=0; i < (imageW + 2*filter_radius) * filter_radius; i++)
				h_Input_padding[i] = 0;
			
			int p = 0, k = 0;
			for (i = (imageW + 2*filter_radius) * filter_radius; i < (imageW + 2*filter_radius) * (filter_radius + imageW); i++ ) {
				if ( p < filter_radius || p >= filter_radius + imageW ) {
					h_Input_padding[i] = 0;
				}
				else {
					h_Input_padding[i] = h_Input[k];
					k++;
				}
				p++;
				if ( p == 2*filter_radius + imageW )
					p = 0;
			}
			
			for (i=(imageW + 2*filter_radius) * (filter_radius + imageW); i < (imageW + 2*filter_radius) * (2*filter_radius + imageW); i++)
				h_Input_padding[i] = 0;
			
			
			


			// To parakatw einai to kommati pou ekteleitai sthn CPU kai me vash auto prepei na ginei h sugrish me thn GPU.
			//printf("CPU computation...\n");

			//clock_gettime(CLOCK_MONOTONIC_RAW, &tv1);

			//convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH, filter_radius); // convolution kata grammes
			//convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, filter_radius); // convolution kata sthles

			//clock_gettime(CLOCK_MONOTONIC_RAW, &tv2);



			//To parakatw einai to kommati pou xreiazetai gia thn ektelesh sthn GPU

			hipEvent_t start, stop;
			hipEventCreate(&start);
			hipEventCreate(&stop);

			dim3 dimGrid1(imageW/TILE_W1, imageH/TILE_H1);
			dim3 dimGrid2(imageW/TILE_W2, imageH/TILE_H2);
			dim3 dimBlock(TILE_WIDTH, TILE_WIDTH);

			

			//Metafora dedomenwn apo ton host pros to device
			gpuErrchk( hipMemcpyToSymbol(HIP_SYMBOL(d_Filter), h_Filter, FILTER_LENGTH * sizeof(float)) );
			gpuErrchk( hipMemcpy(d_Input, h_Input_padding, (imageW + 2*filter_radius) * (2*filter_radius + imageW) * sizeof(float), hipMemcpyHostToDevice) );
			gpuErrchk( hipMemset(d_Buffer, 0, (imageW + 2*filter_radius) * (2*filter_radius + imageW) * sizeof(float)) );

			hipEventRecord(start);

			//Kaloume ton prwto Kernel
			convolutionRowGPU<<<dimGrid1, dimBlock>>>(d_Buffer, d_Input, imageW, imageH);

			gpuErrchk( hipPeekAtLastError() );


			//Kaloume ton deutero Kernel
			convolutionColumnGPU<<<dimGrid2, dimBlock>>>(d_Output, d_Buffer, imageW, imageH);

			gpuErrchk( hipPeekAtLastError() );

			hipEventRecord(stop);
			hipEventSynchronize(stop);
			float GPUtime;
			hipEventElapsedTime(&GPUtime, start, stop);

			//Metafora apotelesmatos apo to device ston host
			gpuErrchk( hipMemcpy(h_OutputGPU, d_Output,  imageW * imageH * sizeof(float), hipMemcpyDeviceToHost) );


			// Kanete h sugrish anamesa se GPU kai CPU kai an estw kai kapoio apotelesma xeperna thn akriveia
			// pou exoume orisei, tote exoume sfalma kai mporoume endexomenws na termatisoume to programma mas

			/*for (i = 0; i < imageW * imageH; i++) {
				if (ABS(h_OutputCPU[i] - h_OutputGPU[i]) > accuracy) {
					printf("Sfalma akriveias \n Termatismos programmatos...\n");
					return(2);
				}
				//printf("h_OutputCPU[%d]: %f\nh_OutputGPU[%d]: %f\n\n", i, h_OutputCPU[i], i, h_OutputGPU[i]);
			}
			*/


			//Ektypwsh xronwn
			/*printf ("Time for the CPU: %10g s\n",
					(double) (tv2.tv_nsec - tv1.tv_nsec) / 1000000000.0 +
					(double) (tv2.tv_sec - tv1.tv_sec));
			printf("Time for the GPU: %f s\n", GPUtime / 1000 );
			*/


			sum_GPU += GPUtime / 1000;
			
			// free all the allocated memory
			hipFree(d_Input);
			hipFree(d_Buffer);
			hipFree(d_Output);

			free(h_OutputGPU);
			//free(h_OutputCPU);
			free(h_Buffer);
			free(h_Input);
			free(h_Filter);
			free(h_Input_padding);
			

			// Do a device reset just in case... Bgalte to sxolio otan ylopoihsete CUDA
			hipDeviceReset();

			printf("End of run %d\n", j);
		}
		
		fprintf(ofp, "GPU: %f\n----------------------\n", sum_GPU/12);

	}

    return 0;
}